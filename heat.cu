#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// Constants
const double DiffusionConstant = 0.25;    // Diffusion constant controlling the rate of heat transfer
const int ImageWidth = 200;               // Width of the simulated image
const int ImageHeight = 200;              // Height of the simulated image
const int BlockSize = 512;                // Number of threads per block in GPU kernel
const int NumIterations = 5000;           // Number of simulation iterations

using namespace std;

// GPU kernel to update temperature values based on the diffusion equation
__global__ void UpdateTemperature(double* next_temp, double* current_temp, int width, int height)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = width * height;

    if (index < total_elements)
    {
        // Update temperature values using the diffusion equation
        next_temp[index] = (1 - 4 * DiffusionConstant) * current_temp[index];
        // Set x and y values for current 
        int x = index % width;
        int y = index / width;

        // Update temperature based on neighboring cells using an approximation of the laplacian using a central difference
        if (x >= 1 && y >= 1) next_temp[index] += DiffusionConstant * current_temp[index - width - 1];
        if (x <= width - 2 && y >= 1) next_temp[index] += DiffusionConstant * current_temp[index - width + 1];
        if (x >= 1 && y <= height - 2) next_temp[index] += DiffusionConstant * current_temp[index + width - 1];
        if (x <= width - 2 && y <= height - 2) next_temp[index] += DiffusionConstant * current_temp[index + width + 1];

        index += blockDim.x * gridDim.x;
    }
}

// GPU kernel to convert temperature values to color
__global__ void TemperatureToColor(unsigned char* color_array, const double* temp_array, int width, int height) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < width * height) {
        // Normalize temperature values and map to RGB color
        double temperature = temp_array[index];
        double normalizedTemp = (temperature - 30000) / (1000 - 30000);
        normalizedTemp = fmax(0.0, fmin(normalizedTemp, 1.0));

        unsigned char r = 255;
        unsigned char g = static_cast<unsigned char>(255 * normalizedTemp);
        unsigned char b = static_cast<unsigned char>(255 * normalizedTemp);

        // Set RGB values in the color array
        color_array[index * 3] = r;
        color_array[index * 3 + 1] = g;
        color_array[index * 3 + 2] = b;
    }
}

int main() 
{
    // Host and device arrays for temperature and color data
    unsigned char* color_array = new unsigned char[ImageWidth * ImageHeight * 3];
    double* temp_array = new double[ImageWidth * ImageHeight];
    double* device_temp_array_current;
    double* device_temp_array_next;
    unsigned char* device_color_array;

    // Allocate device memory for temperature and color arrays
    hipMalloc(&device_color_array, sizeof(unsigned char) * ImageWidth * ImageHeight * 3);
    hipMalloc(&device_temp_array_current, sizeof(double) * ImageWidth * ImageHeight);
    hipMalloc(&device_temp_array_next, sizeof(double) * ImageWidth * ImageHeight);

    // Initialize temperature array with initial conditions
    fill_n(temp_array, ImageWidth * ImageHeight, 1000);
    for (int y = 80; y < 120; ++y)
        for (int x = 80; x < 120; ++x)
            temp_array[y * ImageWidth + x] = 40000;

    // Copy initial temperature array to the device
    hipMemcpy(device_temp_array_current, temp_array, sizeof(double) * ImageWidth * ImageHeight, hipMemcpyHostToDevice);

    // Simulation loop
    for (int i = 0; i < NumIterations; ++i) 
    {
        // Update temperature values on the GPU
        UpdateTemperature<<<(ImageWidth * ImageHeight + BlockSize - 1) / BlockSize, BlockSize>>>(
            i % 2 == 0 ? device_temp_array_next : device_temp_array_current,
            i % 2 == 0 ? device_temp_array_current : device_temp_array_next, ImageWidth, ImageHeight);

        // Convert temperature values to color on the GPU
        TemperatureToColor<<<(ImageWidth * ImageHeight + BlockSize - 1) / BlockSize, BlockSize>>>(
            device_color_array, i % 2 == 0 ? device_temp_array_next : device_temp_array_current, ImageWidth, ImageHeight);

        // Copy color data from the device to the host
        hipMemcpy(color_array, device_color_array, sizeof(unsigned char) * ImageWidth * ImageHeight * 3, hipMemcpyDeviceToHost);
        
        // Save the frame as a PNG image on the host
        char filename[128];
        sprintf(filename, "frame_%04d.png", i);
        stbi_write_png(filename, ImageWidth, ImageHeight, 3, color_array, ImageWidth * 3);
    }

    // Clean up allocated memory
    delete[] color_array;
    delete[] temp_array;
    hipFree(device_color_array);
    hipFree(device_temp_array_current);
    hipFree(device_temp_array_next);

    return 0;
}
